#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <mpi.h>
int* d;




__constant__ int cuda_bf;
__constant__ int cuda_total_vertex;
__constant__ int cuda_tempVertex;
__constant__ int cuda_device_num;
__constant__ int cuda_FW_block;

#define INF 1e9
#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define D2D hipMemcpyDeviceToDevice

using namespace std;

int
init_device ()
{	
	hipSetDevice(0);
	return 0;
}

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
} while (0)

//extern __shared__ int D[];
__global__ void floyd_warshall_1(int* dist,int k ,int kbf){

	int idx,idy;


	idx = k ;
	idy = k ;

	int i = cuda_bf * idx + threadIdx.y;
	int j = cuda_bf * idy + threadIdx.x;
	if(i>=cuda_total_vertex||j>=cuda_total_vertex)
		return ;

	__shared__ int D[32*32];
	D[threadIdx.y*cuda_bf + threadIdx.x] = dist[i*cuda_tempVertex + j];
	__syncthreads();
	// Put to shared memory???
	int x = 0;

	//int dij = dist[i*total_vertex + j];
	//int dik = dist[i*total_vertex + k];
	//int dkj = dist[k*total_vertex + j];
	int dij ,dik,dkj;
	int a = threadIdx.y * cuda_bf + threadIdx.x;
	int b = threadIdx.y * cuda_bf;
	while( x < cuda_bf ){
		dij = D[a];
		dik = D[b + x];
		dkj = D[x*cuda_bf + threadIdx.x];
		if(dij>dik+dkj){
			D[a] = dik + dkj;
		}
		__syncthreads();
		x++;
	}

	dist[i*cuda_tempVertex + j] = D[threadIdx.y*cuda_bf + threadIdx.x];

	return ;
}

__global__ void floyd_warshall_2(int* dist,int k , int kbf  ){

	int idx,idy;

	if(blockIdx.x % 2 == 0 ){
		idx = (blockIdx.x/2) >= k ? (blockIdx.x/2+1):(blockIdx.x/2);
		idy = k;
	}
	else {
		idx = k;
		idy = (blockIdx.x/2) >= k ? (blockIdx.x/2+1):(blockIdx.x/2);
	}

	int i = cuda_bf * idx + threadIdx.y;
	int j = cuda_bf * idy + threadIdx.x;
	//bool flag = 0;
	//if(i>=cuda_total_vertex||j>=cuda_total_vertex)
	//	return;

	__shared__ int D2[32*32*2];
	D2[threadIdx.y * cuda_bf + threadIdx.x] = dist[i*cuda_tempVertex + j];
	D2[(cuda_bf*cuda_bf) + (threadIdx.y *cuda_bf ) + (threadIdx.x)] = dist[ (kbf+threadIdx.y) * cuda_tempVertex + (kbf +threadIdx.x)];
	__syncthreads();
	// Put to shared memory???
	int x = 0;

	int dij ,dik,dkj;
	int a = (threadIdx.y * cuda_bf + threadIdx.x);
	int b;
	if(blockIdx.x%2==0){
		b = cuda_bf*cuda_bf + threadIdx.x;
	}
	else{
		b = cuda_bf*cuda_bf + cuda_bf*threadIdx.y;
	}

	dij = D2[a];

	while(x<cuda_bf){

		if(blockIdx.x%2==0){
			dik = D2[cuda_bf*threadIdx.y + x];
			dkj = D2[b + (x*cuda_bf)];
		}
		else{
			dik = D2[b + x];
			dkj = D2[x*cuda_bf + threadIdx.x];
		}
		if(dij>dik+dkj){
			dij = dik + dkj;
		}
		__syncthreads();
		x++;
	}
	dist[i*cuda_tempVertex + j] = dij;

	return ;
}

__global__ void floyd_warshall_3(int* dist, int k ,int kbf,int ID){

	int idx,idy;

	int blockIdx_x = ((cuda_FW_block-1)/cuda_device_num)*ID + blockIdx.x;


	idy = blockIdx.y >= k? blockIdx.y + 1 : blockIdx.y;
	idx = blockIdx_x >= k? blockIdx_x + 1 : blockIdx_x;

	int i = cuda_bf * idx + threadIdx.y;
	int j = cuda_bf * idy + threadIdx.x;
	//if(i>=cuda_total_vertex||j>=cuda_total_vertex)
	//	return ;

	__shared__ int D3[32*32*3];
	D3[threadIdx.y * cuda_bf + threadIdx.x] = dist[i*cuda_tempVertex + j];
	D3[(cuda_bf*cuda_bf) + (threadIdx.y*cuda_bf) + threadIdx.x] = dist[(cuda_bf*idx+threadIdx.y)*cuda_tempVertex + (kbf + threadIdx.x)];
	D3[(2*cuda_bf*cuda_bf) + (threadIdx.y*cuda_bf) + threadIdx.x] = dist[(kbf+threadIdx.y)*cuda_tempVertex + (idy*cuda_bf+threadIdx.x)];
	__syncthreads();

	// Put to shared memory???
	int x = 0;
	int dij ,dik,dkj;

	int a =threadIdx.y * cuda_bf + threadIdx.x;
	int b = cuda_bf*cuda_bf  + threadIdx.y*cuda_bf;
	int c = 2*cuda_bf*cuda_bf + threadIdx.x; 
	
	dij = D3[a];

	while(x<cuda_bf){
		dik = D3[b + x];
		dkj = D3[x*cuda_bf + c];
		if(dij>dik+dkj){
			dij = dik + dkj;
		}
		x++;
	}
	dist[i*cuda_tempVertex + j] = dij;

	return ;
}



__global__ void floyd_warshall_beta_1(int* dist, int k , int kbf  ){

	int idx,idy;
	idx = k;
	idy = k;
	int i = cuda_bf * idx + (blockIdx.x%cuda_bf);
	int j = cuda_bf * idy + threadIdx.x;
		if(i>=cuda_total_vertex||j>=cuda_total_vertex)
		return ;	

	// Put to shared memory???

	int dij = dist[i*cuda_tempVertex + j];
	int dik = dist[i*cuda_tempVertex + kbf];
	int dkj = dist[kbf*cuda_tempVertex + j];

	if(dij>dik+dkj){
		dist[i*cuda_tempVertex+j] = dik + dkj;
	}

	return ;
}

__global__ void floyd_warshall_beta_2(int* dist, int k , int kbf  ){

	int idx,idy;
	int temp = blockIdx.x / cuda_bf;
	if( (temp) % 2 == 0 ){
		idx = (temp/2) >= k ? (temp/2+1):(temp/2);
		idy = k;
	}
	else {
		idx = k;
		idy = (temp/2) >= k ? (temp/2+1):(temp/2);
	}

	int i = cuda_bf * idx + (blockIdx.x%cuda_bf);
	int j = cuda_bf * idy + threadIdx.x;
		if(i>=cuda_total_vertex||j>=cuda_total_vertex)
		return ;	

	// Put to shared memory???

	int dij = dist[i*cuda_tempVertex + j];
	int dik = dist[i*cuda_tempVertex + kbf];
	int dkj = dist[kbf*cuda_tempVertex + j];

	if(dij>dik+dkj){
		dist[i*cuda_tempVertex+j] = dik + dkj;
	}

	return ;
}

__global__ void floyd_warshall_beta_3(int* dist, int k , int kbf ,int grid_size,int ID ){

	int idx,idy;

	int blockIdx_y = ((cuda_FW_block-1)/cuda_device_num)*ID + blockIdx.y;
	int temp = ((blockIdx_y*gridDim.x) + blockIdx.x) / cuda_bf;
	idx = temp/grid_size  >= k?  temp/grid_size + 1 : temp/grid_size;
	idy = temp % grid_size >= k? temp%grid_size + 1 : temp % grid_size;

	int i = cuda_bf * idx + (blockIdx.x%cuda_bf);
	

	int j = cuda_bf * idy + threadIdx.x;
		if(i>=cuda_total_vertex||j>=cuda_total_vertex)
		return ;	

	// Put to shared memory???

	int x = kbf + cuda_bf;
	int dij ,dik,dkj;

	while(kbf<x){
		dij = dist[i*cuda_tempVertex + j];
		dik = dist[i*cuda_tempVertex + kbf];
		dkj = dist[kbf*cuda_tempVertex + j];
		if(dij>dik+dkj){
			dist[i*cuda_tempVertex + j] = dik + dkj;
		}
		//__syncthreads();
		kbf++;
	}
	return;

}




int main(int argc,char* argv[]){

	hipEvent_t total_start, total_stop;
    hipEvent_t com_start, com_stop;
    hipEvent_t mem_start, mem_stop;
	hipEvent_t io_start, io_stop;
	MPI_Status status;
	MPI_Request req;

	float total_temp=0,total_total=0,io_temp =0 , io_total=0 , com_temp =0,com_total=0 , mem_temp=0 , mem_total=0;

	int rc = MPI_Init(&argc,&argv);
	int rank , process_num;
	if(rc!= MPI_SUCCESS){
		printf("Error when initializing mpi \n");
  	}
  	MPI_Comm_size(MPI_COMM_WORLD,&process_num);
  	MPI_Comm_rank(MPI_COMM_WORLD,&rank);

  	/*
  	if(rank==0){
		hipEventCreate(&total_start);
		hipEventCreate(&total_stop);
		hipEventCreate(&com_start);
		hipEventCreate(&com_stop);
		hipEventCreate(&mem_start);
		hipEventCreate(&mem_stop);
		hipEventCreate(&io_start);
		hipEventCreate(&io_stop);
	}
*/

	hipSetDevice(rank);
	cudaCheckErrors("???");

	//if(rank==0)
	//hipEventRecord(total_start); 

//
	//struct hipDeviceProp_t prop;
	//hipGetDeviceProperties(&prop,0);
	//fprintf(stderr,"clock rate %lf\n",prop.clockRate);

	int bf = atoi(argv[3]);
	int total_vertex;
	int edge_num;

	int DEVICE_NUM = 2;
	int tempVertex;
	int * graph;// = new int[(tempVertex)*(tempVertex)];
	ifstream input;
	ofstream output;
	fprintf(stderr,"IM here\n");	
	if(rank==0){
		
		input.open(argv[1]);

		input >> total_vertex;
		input >> edge_num;

		tempVertex = total_vertex % bf ?  (total_vertex + (bf - (total_vertex%bf) )): total_vertex;
		graph = new int[(tempVertex)*(tempVertex)];

		for(int i=0;i<tempVertex;i++){
			for(int j=0;j<tempVertex;j++){
				graph[i*tempVertex+j] = INF;
			}
			graph[i*tempVertex + i ]=0;
		}

		
		//hipEventRecord(io_start);
		//cudaCheckErrors("4");
		for(int i=0;i<edge_num;i++){
			int a,b;
			input >> a;
			input >> b;
			input >> graph[(a-1)*tempVertex + (b-1) ];
			//fprintf(stderr,"graph %d %d :%d\n",a,b,graph[a*tempVertex+b]);
		}
		MPI_Send(&total_vertex,1,MPI_INT,1,0,MPI_COMM_WORLD);
		MPI_Send(&edge_num,1,MPI_INT,1,0,MPI_COMM_WORLD);
		MPI_Send(&tempVertex,1,MPI_INT,1,0,MPI_COMM_WORLD);

		MPI_Send(graph,tempVertex*tempVertex,MPI_INT,1,0,MPI_COMM_WORLD);

	}
	else{
		MPI_Recv(&total_vertex,1,MPI_INT,0,0,MPI_COMM_WORLD,&status);
		MPI_Recv(&edge_num,1,MPI_INT,0,0,MPI_COMM_WORLD,&status);

		MPI_Recv(&tempVertex,1,MPI_INT,0,0,MPI_COMM_WORLD,&status);
		graph = new int[(tempVertex)*(tempVertex)];
		MPI_Recv(graph,tempVertex*tempVertex,MPI_INT,0,0,MPI_COMM_WORLD,&status);

	}
	fprintf(stderr,"IM there\n");
	//fprintf(stderr,"tempVertex:%d\n",tempVertex);

	//d = new int[tempVertex*tempVertex];
	
	//hipHostMalloc((void**)&graph,sizeof(int)*tempVertex*tempVertex);

	/*
	graph = new int[(tempVertex)*(tempVertex)];

	for(int i=0;i<tempVertex;i++){
		for(int j=0;j<tempVertex;j++){
			graph[i*tempVertex+j] = INF;
		}
		graph[i*tempVertex + i ]=0;
	}

	if(rank==0)
	hipEventRecord(io_start);
	for(int i=0;i<edge_num;i++){
		int a,b;
		input >> a;
		input >> b;
		input >> graph[(a-1)*tempVertex + (b-1) ];
		//fprintf(stderr,"graph %d %d :%d\n",a,b,graph[a*tempVertex+b]);
	}
	*/
	/*
	if(rank==0){
		hipEventRecord(io_stop);
		cudaCheckErrors("1");
		hipEventSynchronize(io_stop);
		cudaCheckErrors("2");
		hipEventElapsedTime(&io_temp,io_start,io_stop);
		cudaCheckErrors("3");
		io_total += io_temp;
	}
*/
	int* cuda_graph;
	

	fprintf(stderr,"1111\n");
	hipMalloc((void**)&cuda_graph,sizeof(int)*(tempVertex)*(tempVertex));
	cudaCheckErrors("malloc gpu");
	fprintf(stderr,"2222\n");
	

	int FWblockDim = tempVertex / bf ;

	//hipSetDevice(0);
	if(rank==0){
		//hipEventRecord(mem_start);	
	}
	cudaCheckErrors("oao");
	hipSetDevice(rank);
		
	hipMemcpy(cuda_graph,graph,sizeof(int)*tempVertex*tempVertex ,H2D);
	cudaCheckErrors("memcpy gpu");
			
	hipMemcpyToSymbol(HIP_SYMBOL(cuda_bf),&bf,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(cuda_total_vertex),&total_vertex,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(cuda_tempVertex),&tempVertex,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(cuda_device_num),&DEVICE_NUM,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(cuda_FW_block),&FWblockDim,sizeof(int));
		
	/*
	if(rank==0){
		hipSetDevice(rank);
		hipEventRecord(mem_stop);
		hipEventSynchronize(mem_stop);
		hipEventElapsedTime(&mem_temp,mem_start,mem_stop);
		mem_total += mem_temp;
	}
	*/
	//int FWblockDim = total_vertex%bf ? (total_vertex/bf + 1) : total_vertex/bf;
	//int remainBF = total_vertex%bf? total_vertex%bf : bf ;
	

	dim3 threadStr(bf,bf);
	dim3 blockStr((FWblockDim-1)/DEVICE_NUM,(FWblockDim-1)/DEVICE_NUM);
	dim3 blockStr_mod((FWblockDim-1)%DEVICE_NUM,(FWblockDim-1)%DEVICE_NUM);
	dim3 blockStr2((FWblockDim-1)*bf,FWblockDim-1);
	//if(rank==0)
	//hipEventRecord(com_start);

	int* copy = new int[tempVertex*tempVertex];
	fprintf(stderr,"IM HERE\n");
	if( bf ==20 && edge_num/total_vertex <= 6){	

			//int threadId = omp_get_thread_num();
			//hipSetDevice(threadId);
			int* type = new int[DEVICE_NUM];
		

			for(int K=0;K<FWblockDim;K++){
				
				// Phase 1

				
					int threadId = rank;
					hipSetDevice(threadId);
					printf("K=%d phase1 id=%d\n",K,threadId);
					floyd_warshall_1<<<1,threadStr>>>(cuda_graph,K,K*bf);
					
					cudaCheckErrors("phase 1");

					//hipDeviceSynchronize()
					// Phase 2
					printf("K=%d phase2 id=%d\n",K,threadId);
					
					if(FWblockDim>1){

							floyd_warshall_2<<< ((FWblockDim-1))*2 ,threadStr>>>(cuda_graph,K,K*bf);
							cudaCheckErrors("phase 2 col");

							// Phase 3
							if(threadId!=DEVICE_NUM-1){
								if(((FWblockDim-1)/DEVICE_NUM)*threadId<K&&((FWblockDim-1)/DEVICE_NUM)*(threadId+1)<=K){
									type[threadId] = 0;
								}
								else if(((FWblockDim-1)/DEVICE_NUM)*threadId<K&&((FWblockDim-1)/DEVICE_NUM)*(threadId+1)>K){
									type[threadId] = 1;
								}
								else{
									type[threadId] = 2;
								}
							}
							else{
								if(((FWblockDim-1)/DEVICE_NUM)*threadId<K&&((FWblockDim-1)/DEVICE_NUM)*(threadId) + ((FWblockDim-1)%DEVICE_NUM + (FWblockDim-1)/DEVICE_NUM)<=K){
									type[threadId] = 0;
								}
								else if(((FWblockDim-1)/DEVICE_NUM)*threadId<K&&((FWblockDim-1)/DEVICE_NUM)*(threadId) + ((FWblockDim-1)%DEVICE_NUM+(FWblockDim-1)/DEVICE_NUM)>K){
									type[threadId] = 1;
								}
								else{
									type[threadId] = 2;
								}
							}

									
				
							dim3 Str_normal((FWblockDim-1)/DEVICE_NUM,FWblockDim-1);
							dim3 Str_last((FWblockDim-1)/DEVICE_NUM + ((FWblockDim-1)%DEVICE_NUM), FWblockDim-1);
							printf("K=%d phase3\n",K);
							
							if(threadId==(DEVICE_NUM-1)&&(((FWblockDim-1)%DEVICE_NUM)!=0)){

								floyd_warshall_3<<<Str_last,threadStr>>>(cuda_graph,K,K*bf,threadId);
								cudaCheckErrors("phase 3 last");	
							}
							else if((FWblockDim-1)/DEVICE_NUM!=0){
								floyd_warshall_3<<<Str_normal,threadStr>>>(cuda_graph,K,K*bf,threadId);
								cudaCheckErrors("phase 3 normal");	
							}
							

					}

				

				if(FWblockDim>1){

					if(rank==0){
						hipSetDevice(rank);
						hipEventRecord(com_stop);
						hipEventSynchronize(com_stop);
						hipEventElapsedTime(&com_temp,com_start,com_stop);
						com_total += com_temp;
						hipEventRecord(mem_start);
					}
					int offset,count;
					
					int i = rank;
								if(type[i]==2){
									offset = tempVertex*((FWblockDim-1)/DEVICE_NUM*bf )*i  + tempVertex*bf;
								}
								else{
									offset = tempVertex*((FWblockDim-1)/DEVICE_NUM*bf )*i ;
								}

								if(i != DEVICE_NUM-1){
									count =  tempVertex*sizeof(int)*((FWblockDim-1)/DEVICE_NUM*bf) ;
								}
								else{
									count = tempVertex*sizeof(int)*(((FWblockDim-1)/DEVICE_NUM*bf)+((FWblockDim-1)%DEVICE_NUM*bf));
								}
								if(type[i]==1){
									count += tempVertex * bf * sizeof(int);
								}

								hipMemcpy(graph+offset,cuda_graph+offset,count,D2H);
								cudaCheckErrors("memcpy");
							//	fprintf(stderr,"ori count %d : %d\n",i,count);


					for(int j=0;j<DEVICE_NUM;j++){
						if(i==j)
							continue;
						MPI_Isend(&type[i],1,MPI_INT,j,j,MPI_COMM_WORLD,&req);	
					}
					for(int j=0;j<DEVICE_NUM;j++){
						if(i==j)
							continue;
						MPI_Recv(&type[j],1,MPI_INT,j,i,MPI_COMM_WORLD,&status);	
						//fprintf(stderr,"rank %d type%d : %d\n",rank,j,type[j]);
					}
					if(count>0){
						for(int j=0;j<DEVICE_NUM;j++){
							if(i==j)
								continue;
							MPI_Isend(&graph[offset],count/sizeof(int),MPI_INT,j,j,MPI_COMM_WORLD,&req);
						}
					}
								
					for(int j=0;j<DEVICE_NUM;j++){
								
								if(i==j)
									continue;

								fprintf(stderr,"%d %d\n",i,j);
								if(type[j]==2){
									offset = tempVertex*((FWblockDim-1)/DEVICE_NUM*bf )*j  + tempVertex*bf;
								}
								else{
									offset = tempVertex*((FWblockDim-1)/DEVICE_NUM*bf )*j ;
								}

								//fprintf(stderr,"OAO\n",i,j);
								if(j != DEVICE_NUM-1){
									count =  tempVertex*sizeof(int)*((FWblockDim-1)/DEVICE_NUM*bf) ;
								}
								else{
									count = tempVertex*sizeof(int)*(((FWblockDim-1)/DEVICE_NUM*bf)+((FWblockDim-1)%DEVICE_NUM*bf));
								}
								if(type[j]==1){
									count += tempVertex * bf * sizeof(int);
								}
								
								//fprintf(stderr,"i:%d j:%d offset:%d count:%d  addi%d   addoff%d  typei:%d  typej:%d \n",i,j,offset,count,cuda_graph[i],cuda_graph[i]+offset,type[i],type[j]);
								if(count>0){
									MPI_Recv(&graph[offset],count/sizeof(int),MPI_INT,j,i,MPI_COMM_WORLD,&status);
								}
								//fprintf(stderr,"i:%d j:%d offset:%d count:%d  addi%d   addoff%d  typei:%d  typej:%d \n",i,j,offset,count,cuda_graph[i],cuda_graph[i]+offset,type[i],type[j]);
								hipMemcpy(cuda_graph+offset,graph+offset,count,H2D);
								cudaCheckErrors("memcpy");	
								
					}
					//fprintf(stderr, "QQ %d\n",rank );
						
					if(rank==0){
			           
				        hipEventRecord(mem_stop);
				        hipEventSynchronize(mem_stop);
				        hipEventElapsedTime(&mem_temp, mem_start, mem_stop);
						mem_total += mem_temp;
						cudaCheckErrors("mem end");
					}
				}
				if(rank==0){
				hipEventRecord(com_start);
				cudaCheckErrors("com start");	
				}
			}
	}
	else{
			int* type = new int[DEVICE_NUM];
		
			
			for(int K=0;K<FWblockDim;K++){
				// Phase 1
				
					int threadId = rank;
					hipSetDevice(threadId);

					//printf("K=%d phase1\n",K);
					for(int i=0;i<bf;i++){
						floyd_warshall_beta_1<<<bf,bf>>>(cuda_graph,K,K*bf + i);
						cudaCheckErrors("phase 1");
					}


					//printf("K=%d phase2\n",K);
					//Phase 2
					

					if(FWblockDim>1){
						for(int i=0;i<bf;i++){
							floyd_warshall_beta_2<<<(FWblockDim-1)*2*bf,bf>>>(cuda_graph,K,K*bf + i );
							cudaCheckErrors("phase 2 col");
						}
						
							if(threadId!=DEVICE_NUM-1){
									if(((FWblockDim-1)/DEVICE_NUM)*threadId<K&&((FWblockDim-1)/DEVICE_NUM)*(threadId+1)<=K){
										type[threadId] = 0;
									}
									else if(((FWblockDim-1)/DEVICE_NUM)*threadId<K&&((FWblockDim-1)/DEVICE_NUM)*(threadId+1)>K){
										type[threadId] = 1;
									}
									else{
										type[threadId] = 2;
									}
								}
								else{
									if(((FWblockDim-1)/DEVICE_NUM)*threadId<K&&((FWblockDim-1)/DEVICE_NUM)*(threadId) + ((FWblockDim-1)%DEVICE_NUM + (FWblockDim-1)/DEVICE_NUM)<=K){
										type[threadId] = 0;
									}
									else if(((FWblockDim-1)/DEVICE_NUM)*threadId<K&&((FWblockDim-1)/DEVICE_NUM)*(threadId) + ((FWblockDim-1)%DEVICE_NUM+(FWblockDim-1)/DEVICE_NUM)>K){
										type[threadId] = 1;
									}
									else{
										type[threadId] = 2;
									}
								}


						//printf("K=%d phase3\n",K);
						//Phase 3
						dim3 Str_normal((FWblockDim-1)*bf,(FWblockDim-1)/DEVICE_NUM);
						dim3 Str_last((FWblockDim-1)*bf, (FWblockDim-1)/DEVICE_NUM + ((FWblockDim-1)%DEVICE_NUM) );
						if(threadId==(DEVICE_NUM-1)&&(((FWblockDim-1)%DEVICE_NUM)!=0)){
							floyd_warshall_beta_3<<<Str_last,bf>>>(cuda_graph,K,K*bf,FWblockDim-1,threadId);
						}
						else if((FWblockDim-1)/DEVICE_NUM!=0){
							floyd_warshall_beta_3<<<Str_normal,bf>>>(cuda_graph,K,K*bf,FWblockDim-1,threadId);
						}
						
						cudaCheckErrors("phase 3");

					}
				
				
				if(FWblockDim>1){

					if(rank==0){
						/*
						hipSetDevice(rank);
						hipEventRecord(com_stop);
						hipEventSynchronize(com_stop);
						hipEventElapsedTime(&com_temp,com_start,com_stop);
						com_total += com_temp;
						hipEventRecord(mem_start);
						*/
					}
					int offset,count;
					
					int i = rank;
								if(type[i]==2){
									offset = tempVertex*((FWblockDim-1)/DEVICE_NUM*bf )*i  + tempVertex*bf;
								}
								else{
									offset = tempVertex*((FWblockDim-1)/DEVICE_NUM*bf )*i ;
								}

								if(i != DEVICE_NUM-1){
									count =  tempVertex*sizeof(int)*((FWblockDim-1)/DEVICE_NUM*bf) ;
								}
								else{
									count = tempVertex*sizeof(int)*(((FWblockDim-1)/DEVICE_NUM*bf)+((FWblockDim-1)%DEVICE_NUM*bf));
								}
								if(type[i]==1){
									count += tempVertex * bf * sizeof(int);
								}

								hipMemcpy(graph+offset,cuda_graph+offset,count,D2H);
								cudaCheckErrors("memcpy");
							//	fprintf(stderr,"ori count %d : %d\n",i,count);


					for(int j=0;j<DEVICE_NUM;j++){
						if(i==j)
							continue;
						MPI_Isend(&type[i],1,MPI_INT,j,j,MPI_COMM_WORLD,&req);	
					}
					for(int j=0;j<DEVICE_NUM;j++){
						if(i==j)
							continue;
						MPI_Recv(&type[j],1,MPI_INT,j,i,MPI_COMM_WORLD,&status);	
						//fprintf(stderr,"rank %d type%d : %d\n",rank,j,type[j]);
					}
					if(count>0){
						for(int j=0;j<DEVICE_NUM;j++){
							if(i==j)
								continue;
							MPI_Isend(&graph[offset],count/sizeof(int),MPI_INT,j,j,MPI_COMM_WORLD,&req);
						}
					}
								
					for(int j=0;j<DEVICE_NUM;j++){
								
								if(i==j)
									continue;

								//fprintf(stderr,"%d %d\n",i,j);
								if(type[j]==2){
									offset = tempVertex*((FWblockDim-1)/DEVICE_NUM*bf )*j  + tempVertex*bf;
								}
								else{
									offset = tempVertex*((FWblockDim-1)/DEVICE_NUM*bf )*j ;
								}

								//fprintf(stderr,"OAO\n",i,j);
								if(j != DEVICE_NUM-1){
									count =  tempVertex*sizeof(int)*((FWblockDim-1)/DEVICE_NUM*bf) ;
								}
								else{
									count = tempVertex*sizeof(int)*(((FWblockDim-1)/DEVICE_NUM*bf)+((FWblockDim-1)%DEVICE_NUM*bf));
								}
								if(type[j]==1){
									count += tempVertex * bf * sizeof(int);
								}
								
								//fprintf(stderr,"i:%d j:%d offset:%d count:%d  addi%d   addoff%d  typei:%d  typej:%d \n",i,j,offset,count,cuda_graph[i],cuda_graph[i]+offset,type[i],type[j]);
								if(count>0){
									MPI_Recv(&graph[offset],count/sizeof(int),MPI_INT,j,i,MPI_COMM_WORLD,&status);
								}
								//fprintf(stderr,"i:%d j:%d offset:%d count:%d  addi%d   addoff%d  typei:%d  typej:%d \n",i,j,offset,count,cuda_graph[i],cuda_graph[i]+offset,type[i],type[j]);
								hipMemcpy(cuda_graph+offset,graph+offset,count,H2D);
								cudaCheckErrors("memcpy");	
								
					}
					//fprintf(stderr, "QQ %d\n",rank );
						
					if(rank==0){
			           /*
				        hipEventRecord(mem_stop);
				        hipEventSynchronize(mem_stop);
				        hipEventElapsedTime(&mem_temp, mem_start, mem_stop);
						mem_total += mem_temp;
						cudaCheckErrors("mem end");
						*/
					}
				}
				if(rank==0){
					/*
					hipEventRecord(com_start);
					cudaCheckErrors("com start");	
					*/
				}
		}
	}

	MPI_Finalize();


	fprintf(stderr,"IM THERE %d\n",rank);
	//fprintf(stderr,"%d QAQ \n",rank);
//	hipSetDevice(rank);
	//hipDeviceSynchronize();

	// 時間計算是否要擺到前面??
	//if(rank==0){
	//hipEventRecord(com_stop);
	//hipEventSynchronize(com_stop);
	//hipEventElapsedTime(&com_temp,com_start,com_stop);
	//com_total+=com_temp;
//	}
	
	//fprintf(stderr,"%d QAQ 2\n",rank);
	//fprintf(stderr,"qqq %s   %s  \n",typeid((&graph[0])+10),typeid(cuda_graph[1]+10));
	if(rank==0){
		//hipEventRecord(mem_start);
		hipMemcpy(graph,cuda_graph,sizeof(int)*tempVertex*tempVertex,D2H);

		//fprintf(stderr,"%d QAQ QAQ\n",rank);
		//cudaCheckErrors("copy back error");
		//hipEventRecord(mem_stop);
		//hipEventSynchronize(mem_stop);
		//hipEventElapsedTime(&mem_temp,mem_start,mem_stop);
		mem_total += mem_temp;
	}
	cudaCheckErrors("QQQ");
		//fprintf(stderr,"%d QAQ3 \n",rank);
	/*
	#pragma omp parallel num_threads(DEVICE_NUM)
	{

		int* tempGraph = new int[tempVertex*tempVertex];
		int threadId = omp_get_thread_num();

		hipSetDevice(threadId);
		
		int offset = tempVertex*(tempVertex/DEVICE_NUM) * threadId;
		int count = threadId==(DEVICE_NUM-1)? tempVertex*( (tempVertex/DEVICE_NUM) + (tempVertex%DEVICE_NUM)) : tempVertex*(tempVertex/DEVICE_NUM) ;
			


		hipMemcpy(tempGraph,(cuda_graph[threadId]),sizeof(int)*tempVertex*tempVertex,D2H);
		for(int i=offset;i<offset+count;i++){
			graph[i] = tempGraph[i];
		}
		cudaCheckErrors("copy back error");
	

	}
	*/

	if(rank==0){
		//hipEventRecord(io_start);

	
		output.open(argv[2]);
		// 每行最後面到底要不要加SPACE!!!!!!!

		

			for(int i=0;i<total_vertex;i++){

				for(int j=0;j<total_vertex;j++){
					if(graph[i*tempVertex+j]==INF){
						output<<"INF";
					}
					else{
						output<<graph[i*tempVertex+j];
					}
					output<<" ";
				}
				output<<endl;
			}
		
	//
		//hipEventRecord(io_stop);
		//hipEventSynchronize(io_stop);
		//hipEventElapsedTime(&io_temp,io_start,io_stop);
		//io_total += io_temp;
	
	///hipEventRecord(total_stop);
   // hipEventSynchronize(total_stop);
	//hipEventElapsedTime(&total_temp, total_start, total_stop);
	}

	if(rank==0){
	    fprintf(stderr, "\n\n");
	    fprintf(stderr, "TOTAL = %f\n", total_temp);
	    fprintf(stderr, "COMPUTE = %f\n", com_total);
	    fprintf(stderr, "MEMORY = %f\n", mem_total);
		fprintf(stderr, "IO = %f\n", io_total);
	}
	return 0;
}